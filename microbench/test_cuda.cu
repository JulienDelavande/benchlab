// test_cuda.cu
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void square(float* x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    x[idx] *= x[idx];
}

int main() {
    float* d;
    hipMalloc(&d, sizeof(float) * 1024);
    square<<<1, 1024>>>(d);
    hipDeviceSynchronize();
    hipFree(d);
    std::cout << "Done" << std::endl;
    return 0;
}
